/*

This program will numerically compute the integral of

                  4/(1+x*x) 
				  
from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <iostream>
#include <fstream>

static long num_steps = 100000000;
static int num_blocks = 1;
static int num_threads = 1;
double step;

__global__ void piAdd(float* sums, long num_steps){
      
      int tid = blockIdx.x * blockDim.x + threadIdx.x;
      double x, sum = 0.0;
      int stride = blockDim.x * gridDim.x;
      double step = 1.0/(double) num_steps;

      for (int i = tid; i < num_steps; i += stride){
        x = (i-0.5)*step;
        sum = sum + 4.0/(1.0+x*x);
      }

      atomicAdd(sums, sum);

}


int main (int argc, char** argv)
{
    
    // Read command line arguments.
	for (int i = 0; i < argc; i++)
	{
		if ((strcmp(argv[i], "-N") == 0) || (strcmp(argv[i], "-num_steps") == 0))
		{
			num_steps = atol(argv[++i]);
			printf("  User num_steps is %ld\n", num_steps);
		}
		else if ((strcmp(argv[i], "-B") == 0) || (strcmp(argv[i], "-num_blocks") == 0))
		{
			num_blocks = atol(argv[++i]);
			printf("  User num_blocks is %ld\n", num_blocks);
		}
		else if ((strcmp(argv[i], "-T") == 0) || (strcmp(argv[i], "-num_threads") == 0))
		{
			num_threads = atol(argv[++i]);
			printf("  User num_threads is %ld\n", num_threads);
		}
		else if ((strcmp(argv[i], "-h") == 0) || (strcmp(argv[i], "-help") == 0))
		{
			printf("  Pi Options:\n");
			printf("  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n");
			printf("  -num_blocks (-B) <int>:      Number of blocks used to compute Pi (by default 1)\n");
			printf("  -num_threads (-T) <int>:      Number of threads per block to compute Pi (by default 1)\n");
			printf("  -help (-h):            print this message\n\n");
			exit(1);
		}
	}

	double pi = 0.0;
	  
    step = 1.0/(double) num_steps;

    // Allocate host memory
	float *h_sum = (float *)malloc(sizeof(float) * num_blocks);
	for (int i = 0; i < num_blocks; i++)
	{
		h_sum[i] = 0.0;
	}

	// Allocate device memory
	float *d_sum;
	hipMalloc((void **)&d_sum, sizeof(float) * num_blocks);

	hipMemcpy(d_sum, h_sum, sizeof(float) * num_blocks, hipMemcpyHostToDevice);

    // Timer products.
    struct timeval begin, end;

    gettimeofday( &begin, NULL );

    piAdd<<<num_blocks, num_threads>>>(d_sum, num_steps);
	hipDeviceSynchronize();
	hipMemcpy(h_sum, d_sum, sizeof(float) * num_blocks, hipMemcpyDeviceToHost);

    pi = *h_sum * step;


    gettimeofday( &end, NULL );

    // Calculate time.
    double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
            1.0e-6 * ( end.tv_usec - begin.tv_usec );
            
    printf("\n pi with %ld steps is %lf in %lf seconds\n ",num_steps,pi,time);

    std::fstream output;
	output.open("pi_stats.csv", std::ios_base::app);
	output << "very basic"
		   << ", " << num_blocks << ", " << num_threads << ", " << num_steps << ", " << time << "\n";
}
